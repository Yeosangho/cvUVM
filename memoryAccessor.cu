#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;
void cuSetDeviceFlags(){
	hipSetDeviceFlags(hipDeviceMapHost);
}
void cuMallocManaged(void** h_img, int r, int c){

	hipMallocManaged(h_img,sizeof(unsigned char)*r*c);

}

void cuMalloc(void** h_img, int r, int c){
	hipMalloc(h_img, sizeof(float)*r*c);
}

void cuDeviceSynchronize(){
	hipDeviceSynchronize();
}
