#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;
void cuSetDeviceFlags(){
	hipSetDeviceFlags(hipDeviceMapHost);
}
void cuMallocManaged(float** h_img, int r, int c){

	hipMallocManaged(h_img,sizeof(float)*r*c);

}

void cuMalloc(void** h_img, int r, int c){
	hipMalloc(h_img, sizeof(float)*r*c);
}

void cuDeviceSynchronize(){
	hipDeviceSynchronize();
}
